#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
// Dont forget to add cuda_runtime header file
#include <hip/hip_runtime.h>

constexpr unsigned int N = 5;

__global__ void kernel_expression_evaluation_1(int* ret_01_v, const int* a, const int* b, const int* c, const int* d)
{
    const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("tid=%d, blockIdx.x=%d, threadIdx.x=%d\n", tid, blockIdx.x, threadIdx.x);
    printf("a[%d]=%d, b[%d]=%d, c[%d]=%d, d[%d]=%d;\n", tid, a[tid], tid,b[tid], tid,c[tid], tid,d[tid]);
    ret_01_v[tid] = (a[tid] + b[tid] * c[tid] > d[tid]);
    // printf("For thread %d, corresponding value is %d\n", tid, ret_01_v[tid]);
}

// __global__ void kernel_scan_x_column(int* ret_x, const int* ret_01_v, const int* x, const int* psum, unsigned int n)
// {
//     const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     const unsigned int stride = blockDim.x * gridDim.x;
//     // unsigned int local_count = psum[tid];
//     for(unsigned int i = tid; i < n; i += stride)
//     {
//         if(ret_01_v[i] == 1)
//         {
//             ret_x[local_count] = x[i];
//         }
//     }
// }

__global__ void kernel_scan_x_column(int* ret_x, const int* ret_01_v, const int* x, const int* psum, unsigned int n)
{
    const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    for(unsigned int i = tid; i < n; i += stride)
    {
        unsigned int local_count = psum[i] - 1;
        if(ret_01_v[i] == 1)
        {
            ret_x[local_count] = x[i];
        }
    }
}

int main() {
    thrust::host_vector<int> a_host(N);
    for (int i = 0; i < N; i++) {
        a_host[i] = i + 1;
    }
    thrust::device_vector<int> a = a_host;

    thrust::host_vector<int> b_host(N, 1);
    thrust::device_vector<int> b = b_host;

    thrust::host_vector<int> c_host(N, 2);
    thrust::device_vector<int> c = c_host;

    thrust::host_vector<int> d_host(N, 3);
    thrust::device_vector<int> d = d_host;

    thrust::host_vector<int> ret_01_v_host(N);
    thrust::device_vector<int> ret_01_v(N);

    // kernel_expression_evaluation_1<<<1, N>>>(ret_01_v.data(), a.data(), b.data(), c.data(), d.data());
    kernel_expression_evaluation_1<<<1, N>>>(thrust::raw_pointer_cast(ret_01_v.data()),
                                             thrust::raw_pointer_cast(a.data()),
                                             thrust::raw_pointer_cast(b.data()),
                                             thrust::raw_pointer_cast(c.data()),
                                             thrust::raw_pointer_cast(d.data()));
    hipDeviceSynchronize();

    ret_01_v_host = ret_01_v;
    // thrust::copy(ret_01_v.begin(), ret_01_v.end(), ret_01_v_host.begin());
    std::cout << "ret_01_v_host elements are: ";
    for (auto ret_01_v_host_elem: ret_01_v_host) {
        std::cout << ret_01_v_host_elem << " ";
    }
    std::cout << std::endl;

    thrust::device_vector<int> psum(N);
    thrust::inclusive_scan(ret_01_v.begin(), ret_01_v.end(), psum.begin());

    thrust::host_vector<int> psum_host = psum;
    std::cout << "psum_host elements are: ";
    for (auto psum_host_elem: psum_host) {
        std::cout << psum_host_elem << " ";
    }
    std::cout << std::endl;

    thrust::host_vector<int> x_host(N, 10);
    for (int i = 0; i < N; i++) {
        x_host[i] += i + 1;
    }

    thrust::device_vector<int> x = x_host;

    thrust::host_vector<int> ret_x_host;
    thrust::device_vector<int> ret_x(N, 0);
    // thrust::host_vector<int> y_host(N);
    // thrust::host_vector<int> z_host(N);

    kernel_scan_x_column<<<1, 2>>>(thrust::raw_pointer_cast(ret_x.data()),
        thrust::raw_pointer_cast(ret_01_v.data()),
        thrust::raw_pointer_cast(x.data()),
        thrust::raw_pointer_cast(psum.data()),
        N);

    ret_x_host = ret_x;
    std::cout << "ret_x_host elements are: ";
    for (const auto ret_x_host_elem: ret_x_host) {
        std::cout << ret_x_host_elem << " ";
    }
    std::cout << std::endl;
    return 0;
}